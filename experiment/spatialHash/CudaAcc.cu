#include "hip/hip_runtime.h"
#include <OptiX/_Define_7_Device.h>

__global__ void initRandom(hiprandState* state, unsigned int seed, unsigned int MaxNum)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < MaxNum)
		hiprand_init(seed, id, 0, state + id);
}
void initRandom(hiprandState* state, int seed, unsigned int block, unsigned int grid, unsigned int MaxNum)
{
	initRandom << <grid, block >> > (state, seed, MaxNum);
}