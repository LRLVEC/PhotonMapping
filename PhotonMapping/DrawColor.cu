#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;
using namespace Define;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtBuffer<float3>vertexBufferIndexed;
rtBuffer<float3>normalBuffer;
rtBuffer<float3>colorBuffer;
rtBuffer<uint3>indexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, texid, , );
rtDeclareVariable(float3, color, , );
rtDeclareVariable(float, n, , );
rtDeclareVariable(float3, decay, , );
rtDeclareVariable(float, scatter, , );
rtDeclareVariable(float, cosTheta, , );
rtDeclareVariable(Trans, trans, , );
rtDeclareVariable(TransDepth, transDepth, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(unsigned int, depthMax, , );
rtDeclareVariable(unsigned int, glassDepthMax, , );
rtDeclareVariable(unsigned int, russian, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayData, rayData, rtPayload, );
rtDeclareVariable(float, l, rtIntersectionDistance, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, colored, attribute colored, );

//rtTextureSampler<uchar4, 3, hipReadModeNormalizedFloat> ahh;

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	int2 screen = make_int2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray rayOrigin(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	//rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void rayAllocatorDepth()
{
	size_t2 size(result.size());
	int2 screen = make_int2(size.x, size.y);
	float2 ahh = 2.0f * random(index, screen, frame) -
		make_float2(screen.x - int(index.x * 2), screen.y - int(index.y * 2));
	float2 delta(transDepth.D * randomCircle(random1(index, screen, frame)));
	float4 d = make_float4(ahh * transDepth.P - delta, transDepth.V, 0);
	Ray rayOrigin(transDepth.r0 + transDepth.trans * make_float4(delta, 0, 0),
		normalize(transDepth.trans * d), CloseRay, offset);
	RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	//rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void metalAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void metalCloseHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		rayNow.direction = ray.direction - 2 * dot(ray.direction, normal) * normal;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		rtTrace(group, rayNow, rayDataNow);
		answer = rayDataNow.color * color;
	}
	rayData.color = answer * colored;
}
RT_PROGRAM void glassAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void glassCloseHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < glassDepthMax)
	{
		float4 r = make_float4(1);
		float4 t = make_float4(1);
		float cosi1 = dot(ray.direction, normal);
		float nt(cosi1 < 0 ? n : 1 / n);
		float sini1 = sqrtf(1 - cosi1 * cosi1);
		float sini2 = sini1 / nt;
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		if (sini2 < 1)
		{
			float cosi2 = sqrtf(1 - sini2 * sini2);
			if (sini2 <= 0.02)
			{
				float ahh = 4 * nt / ((nt + 1) * (nt + 1));
				t.w *= ahh;
				r.w *= 1 - ahh;
			}
			else
			{
				float a1 = nt * fabsf(cosi1) + cosi2;
				float a2 = fabsf(cosi1) + nt * cosi2;
				r.w *= (pow((nt * cosi2 - fabsf(cosi1)) / a2, 2) + pow((cosi2 - nt * fabsf(cosi1)) / a1, 2)) / 2;
				t.w *= 2 * cosi2 * (1 / pow(a1, 2) + 1 / pow(a2, 2)) * nt * fabsf(cosi1);
			}
			rayNow.direction = (ray.direction + (nt * ::copysignf(cosi2, cosi1) - cosi1) * normal) / nt;
			rayNow.tmin = offset;
			rayNow.tmax = RT_DEFAULT_MAX;
			rtTrace(group, rayNow, rayDataNow);
			if (cosi1 > 0) { float3 s(expf(-decay * l)); *(float3*)& t *= s; *(float3*)& r *= s; }
			answer += rayDataNow.color * make_float3(t) * t.w;
		}
		else
			*(float3*)& r = expf(-decay * l);
		rayNow.direction = ray.direction - 2 * cosi1 * normal;
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * make_float3(r) * r.w;
	}
	rayData.color = answer * colored;
}
RT_PROGRAM void diffuseAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void diffuseCloseHIt()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		float k(1);
		float2 seed(make_float2(ray.origin.y - sqrtf(frame), ray.direction.z + sqrtf(frame)));
		if (rayData.depth > russian)
		{
			if (random(seed) < 0.2f) { rayData.color = answer; return; }
			else k /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		rayNow.direction = randomDirectionCosN(cosi1 <= 0 ? normal : -normal, 1, seed);
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * color * k;
	}
	rayData.color = answer * colored;
}
RT_PROGRAM void scatterAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void scatterCloseHit()
{
	float3 answer = make_float3(0);
	float3 ratio;
	float ratioW(1);
	if (rayData.depth < depthMax)
	{
		float3 seed(make_float3(
			ray.origin.y - sqrtf(frame),
			ray.direction.z + sqrtf(frame),
			ray.direction.x + sqrtf(frame)
		));
		float gg(random(make_float2(seed)));
		if (rayData.depth > russian)
		{
			if (gg < 0.2f) { rayData.color = answer; return; }
			else ratioW /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		gg = random(make_float2(seed.y, seed.z));
		if (cosi1 > 0 && gg > expf(-l * scatter))
		{
			float4 ahh(randomScatter(ray.direction, l, scatter, seed));
			rayNow.origin = ray.origin + ahh.w * ray.direction;
			rayNow.direction = make_float3(ahh);
			ratio = scatterRatio * expf(-decay * ahh.w);
		}
		else
		{
			rayNow.origin = ray.origin + l * ray.direction;
			rayNow.direction = ray.direction;
			if (cosi1 > 0)ratio = expf(-decay * l);
			else ratio = { 1,1,1 };
		}
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * ratio * ratioW;
	}
	rayData.color = answer * colored;
}
RT_PROGRAM void glassScatterAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void glassScatterCloseHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < 20)
	{
		float3 seed(make_float3(
			ray.origin.y - sqrtf(frame),
			ray.direction.z + sqrtf(frame),
			ray.direction.x + sqrtf(frame)
		));
		float gg(random(make_float2(seed)));
		float ratioW(1);
		if (rayData.depth > russian)
		{
			if (gg < 0.2f) { rayData.color = answer; return; }
			else ratioW /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		gg = random(make_float2(seed.y, seed.z));
		if (cosi1 < 0 || gg < expf(-l * scatter))
		{
			float nt(cosi1 < 0 ? n : 1 / n);
			float sini1 = sqrtf(1 - cosi1 * cosi1);
			float sini2 = sini1 / nt;
			float4 r = make_float4(1);
			float4 t = make_float4(1);
			rayNow.origin = ray.origin + l * ray.direction;
			if (sini2 < 1)
			{
				float cosi2 = sqrtf(1 - sini2 * sini2);
				if (sini2 <= 0.02)
				{
					float ahh = 4 * nt / ((nt + 1) * (nt + 1));
					t.w *= ahh;
					r.w *= 1 - ahh;
				}
				else
				{
					float a1 = nt * fabsf(cosi1) + cosi2;
					float a2 = fabsf(cosi1) + nt * cosi2;
					r.w *= (pow((nt * cosi2 - fabsf(cosi1)) / a2, 2) + pow((cosi2 - nt * fabsf(cosi1)) / a1, 2)) / 2;
					t.w *= 2 * cosi2 * (1 / pow(a1, 2) + 1 / pow(a2, 2)) * nt * fabsf(cosi1);
				}
				rayNow.direction = (ray.direction + (nt * ::copysignf(cosi2, cosi1) - cosi1) * normal) / nt;
				rtTrace(group, rayNow, rayDataNow);
				if (cosi1 > 0) { float3 s(expf(-decay * l)); *(float3*)& t *= s; *(float3*)& r *= s; }
				answer += rayDataNow.color * make_float3(t) * t.w;
			}
			else
				*(float3*)& r = expf(-decay * l);
			rayNow.direction = ray.direction - 2 * cosi1 * normal;
			rtTrace(group, rayNow, rayDataNow);
			answer += rayDataNow.color * make_float3(r) * r.w;
			rayData.color = answer * colored;
			return;
		}
		else
		{
			float sini1 = sqrtf(1 - cosi1 * cosi1);
			float sini2 = sini1 / n;
			float cosi2 = sqrtf(1 - sini2 * sini2);
			float4 ahh(randomScatter((ray.direction + (n * ::copysignf(cosi2, cosi1) - cosi1) * normal) / n, l, scatter, seed));
			rayNow.origin = ray.origin + ahh.w * ray.direction;
			rayNow.direction = make_float3(ahh);
			rtTrace(group, rayNow, rayDataNow);
			rayData.color = rayDataNow.color * colored * expf(-decay * ahh.w) * ratioW;
			return;
		}
	}
}
RT_PROGRAM void lightAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void lightCloseHit()
{
	if (dot(normal, ray.direction) < 0)
		rayData.color = color * colored;// *normalize(normal + make_float3(1));
	else
		rayData.color = { 0,0,0 };
}
RT_PROGRAM void torchAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void torchCloseHit()
{
	float a(dot(ray.direction, normal));
	if (a < 0)
	{
		if (a < -cosTheta)rayData.color = color;
		else if (rayData.depth == 0)rayData.color = { 1, 1, 1 };
	}
	else rayData.color = { 0,0,0 };
}
RT_PROGRAM void miss()
{
	rayData.color = make_float3(rtTexCubemap<float4>(texid, ray.direction.x, ray.direction.y, ray.direction.z));
	//float3 source{ 0.9,0.4,-0.6 };
	//float3 dir(normalize(ray.origin - source));
	//if (dir.x < -0.999 && rayData.depth && dot(dir, ray.direction) < -0.9)
	//	rayData.color = { 10, 10, 10 };//
	//else
	//rayData.color = { 0 };
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}
RT_PROGRAM void attrib()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(cross(d1, d2))));
	colored = { 1,1,1 };
}
RT_PROGRAM void attribIndexed()
{
	uint3 id = indexBuffer[rtGetPrimitiveIndex()];
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(
		texcoord.x * normalBuffer[id.y] +
		texcoord.y * normalBuffer[id.z] +
		(1 - texcoord.x - texcoord.y) * normalBuffer[id.x])));
	colored = { 1,1,1 };
}
RT_PROGRAM void attribColored()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(cross(d1, d2))));
	colored = colorBuffer[id];
}